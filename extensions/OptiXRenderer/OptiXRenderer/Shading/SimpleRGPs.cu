#include "hip/hip_runtime.h"
// Simple OptiX ray generation programs, such as path tracing, normal and albedo visualization
// ------------------------------------------------------------------------------------------------
// Copyright (C) Bifrost. See AUTHORS.txt for authors.
//
// This program is open source and distributed under the New BSD License.
// See LICENSE.txt for more detail.
// ------------------------------------------------------------------------------------------------

#include <OptiXRenderer/Shading/ShadingModels/DefaultShading.h>
#include <OptiXRenderer/Shading/LightSources/LightImpl.h>
#include <OptiXRenderer/Types.h>
#include <OptiXRenderer/Utils.h>

#include <optix.h>
#include <optixu/optixu_matrix_namespace.h>

using namespace OptiXRenderer;
using namespace optix;

rtDeclareVariable(uint2, g_launch_index, rtLaunchIndex, );

rtDeclareVariable(CameraStateGPU, g_camera_state, , );
rtBuffer<Material, 1> g_materials;

// ------------------------------------------------------------------------------------------------
// Ray generation program utility functions.
// ------------------------------------------------------------------------------------------------

// Scene variables
rtDeclareVariable(rtObject, g_scene_root, , );
rtDeclareVariable(SceneStateGPU, g_scene, , );

const int pmj_period = 9;
__constant__ float2 pmj_offsets[81] = {
    { 0.93f, 0.43f }, { 0.48f, 0.38f }, { 0.96f, 0.77f }, { 0.22f, 0.02f }, { 0.23f, 0.14f }, { 0.95f, 0.65f }, { 0.67f, 0.07f }, { 0.06f, 0.56f }, { 0.99f, 0.99f },
    { 0.51f, 0.60f }, { 0.27f, 0.47f }, { 0.88f, 0.98f }, { 0.72f, 0.52f }, { 0.52f, 0.72f }, { 0.20f, 0.79f }, { 0.53f, 0.83f }, { 0.16f, 0.46f }, { 0.09f, 0.78f },
    { 0.78f, 0.09f }, { 0.33f, 0.04f }, { 0.36f, 0.26f }, { 0.80f, 0.31f }, { 0.14f, 0.23f }, { 0.69f, 0.30f }, { 0.57f, 0.17f }, { 0.12f, 0.12f }, { 0.58f, 0.28f },
    { 0.84f, 0.64f }, { 0.32f, 0.91f }, { 0.31f, 0.80f }, { 0.85f, 0.75f }, { 0.35f, 0.15f }, { 0.73f, 0.63f }, { 0.07f, 0.67f }, { 0.04f, 0.33f }, { 0.28f, 0.58f },
    { 0.64f, 0.84f }, { 0.70f, 0.41f }, { 0.15f, 0.35f }, { 0.68f, 0.19f }, { 0.89f, 0.10f }, { 0.30f, 0.69f }, { 0.77f, 0.96f }, { 0.11f, 0.01f }, { 0.10f, 0.89f },
    { 0.91f, 0.32f }, { 0.75f, 0.85f }, { 0.25f, 0.25f }, { 0.83f, 0.53f }, { 0.56f, 0.06f }, { 0.17f, 0.57f }, { 0.60f, 0.51f }, { 0.42f, 0.81f }, { 0.26f, 0.36f },
    { 0.37f, 0.37f }, { 0.21f, 0.90f }, { 0.19f, 0.68f }, { 0.41f, 0.70f }, { 0.63f, 0.73f }, { 0.90f, 0.21f }, { 0.47f, 0.27f }, { 0.86f, 0.86f }, { 0.05f, 0.44f },
    { 0.59f, 0.40f }, { 0.74f, 0.74f }, { 0.02f, 0.22f }, { 0.40f, 0.59f }, { 0.46f, 0.16f }, { 0.01f, 0.11f }, { 0.94f, 0.54f }, { 0.43f, 0.93f }, { 0.00f, 0.00f },
    { 0.49f, 0.49f }, { 0.44f, 0.05f }, { 0.98f, 0.88f }, { 0.79f, 0.20f }, { 0.62f, 0.62f }, { 0.81f, 0.42f }, { 0.38f, 0.48f }, { 0.65f, 0.95f }, { 0.54f, 0.94f },
};

__inline_dev__ MonteCarloPayload initialize_monte_carlo_payload(int x, int y, int image_width, int image_height,
    int accumulation_count, optix::float3 camera_position, const optix::Matrix4x4& inverted_view_projection_matrix) {
    using namespace optix;

    MonteCarloPayload payload;
    payload.radiance = make_float3(0.0f);

    RNG::LinearCongruential pmj_offset_rng; pmj_offset_rng.seed(__brev(RNG::teschner_hash(x, y)));
    float2 offset = pmj_offsets[(x % pmj_period) + (y % pmj_period) * pmj_period];
    offset += make_float2(-1 / 18.0f) + pmj_offset_rng.sample2f() / pmj_period;
    payload.pmj_rng_state = PMJSamplerState::make(accumulation_count, offset.x, offset.y);

    /*
    const int period = 5;
    const float pixel_span = 1.0f / period;
    int y_stratum = (x % period);
    y_stratum ^= y_stratum >> 1;
    float pmj_y_offset = y_stratum * pixel_span + pixel_span * pmj_offset_rng.sample1f();
    int x_stratum = ((x + y) % period);
    x_stratum ^= x_stratum >> 1;
    float pmj_x_offset = x_stratum * pixel_span + pixel_span * pmj_offset_rng.sample1f();
    payload.pmj_rng_state = PMJSampler::make(accumulation_count, pmj_x_offset, pmj_y_offset);
    */

    payload.throughput = make_float3(1.0f);
    payload.bounces = 0;
    payload.bsdf_MIS_PDF = 0.0f;
    payload.shading_normal = make_float3(0.0f);
    payload.material_index = 0;

    // Generate rays.
    RNG::LinearCongruential rng; rng.seed(__brev(RNG::teschner_hash(x, y, accumulation_count)));
    float2 screen_pos = make_float2(x, y) + (accumulation_count == 0 ? make_float2(0.5f) : rng.sample2f());
    float2 viewport_pos = make_float2(screen_pos.x / float(image_width), screen_pos.y / float(image_height));
    payload.position = camera_position;
    payload.direction = project_ray_direction(viewport_pos, payload.position, inverted_view_projection_matrix);
    return payload;
}

template <typename Evaluator>
__inline_dev__ void accumulate(Evaluator evaluator) {
    const CameraStateGPU& camera_state = g_camera_state;
    const int accumulation_count = camera_state.accumulations;
    size_t2 screen_size = camera_state.accumulation_buffer.size();

    MonteCarloPayload payload = initialize_monte_carlo_payload(g_launch_index.x, g_launch_index.y,
        screen_size.x, screen_size.y, accumulation_count,
        camera_state.camera_position, camera_state.inverted_view_projection_matrix);

    float3 radiance = evaluator(payload);

    auto accumulation_buffer = camera_state.accumulation_buffer;
#ifdef DOUBLE_PRECISION_ACCUMULATION_BUFFER
    double3 accumulated_radiance_d;
    if (accumulation_count != 0) {
        double3 prev_radiance = make_double3(accumulation_buffer[g_launch_index].x, accumulation_buffer[g_launch_index].y, accumulation_buffer[g_launch_index].z);
        accumulated_radiance_d = lerp_double(prev_radiance, make_double3(radiance.x, radiance.y, radiance.z), 1.0 / (accumulation_count + 1.0));
    } else
        accumulated_radiance_d = make_double3(radiance.x, radiance.y, radiance.z);
    accumulation_buffer[g_launch_index] = make_double4(accumulated_radiance_d.x, accumulated_radiance_d.y, accumulated_radiance_d.z, 1.0f);
    float3 accumulated_radiance = make_float3(accumulated_radiance_d.x, accumulated_radiance_d.y, accumulated_radiance_d.z);
#else
    float3 accumulated_radiance;
    if (accumulation_count != 0) {
        float3 prev_radiance = make_float3(accumulation_buffer[g_launch_index]);
        accumulated_radiance = lerp(prev_radiance, radiance, 1.0f / (accumulation_count + 1.0f));
    }
    else
        accumulated_radiance = radiance;
    accumulation_buffer[g_launch_index] = make_float4(accumulated_radiance, 1.0f);
#endif

    camera_state.output_buffer[g_launch_index] = float_to_half(make_float4(accumulated_radiance, 1.0f));
}

//-------------------------------------------------------------------------------------------------
// Path tracing ray generation program.
//-------------------------------------------------------------------------------------------------
RT_PROGRAM void path_tracing_RPG() {

    accumulate([](MonteCarloPayload payload) -> float3 {
        do {
            Ray ray(payload.position, payload.direction, RayTypes::MonteCarlo, g_scene.ray_epsilon);
            rtTrace(g_scene_root, ray, payload);
        } while (payload.bounces < g_camera_state.max_bounce_count && !is_black(payload.throughput));

        return payload.radiance;
    });
}

//-------------------------------------------------------------------------------------------------
// Denoise ray generation program.
//-------------------------------------------------------------------------------------------------
namespace AIDenoiser {

rtDeclareVariable(AIDenoiserStateGPU, g_AI_denoiser_state, , );

RT_PROGRAM void path_tracing_RPG() {
    float3 albedo = { 0, 0, 0 };
    float3 normal = { 0, 0, 0 };

    accumulate([&](MonteCarloPayload payload) -> float3 {
        bool properties_accumulated = false;
        do {
            float3 last_ray_direction = payload.direction;
            Ray ray(payload.position, payload.direction, RayTypes::MonteCarlo, g_scene.ray_epsilon);
            rtTrace(g_scene_root, ray, payload);

            bool terminate_ray = !(payload.bounces < g_camera_state.max_bounce_count && !is_black(payload.throughput));

            // Accumulate surface properties of the first non-specular surface hit.
            // If no non-specular hits are found or the BSDF PDF is zero due to a bad sample being drawn, 
            // then use the last hit to ensure that some feature data is output.
            if (!properties_accumulated && (payload.bsdf_MIS_PDF != 0.0f || terminate_ray)) {
                properties_accumulated = true;

                normal = payload.shading_normal;

                if (payload.material_index > 0) {
                    using namespace Shading::ShadingModels;
                    const float abs_cos_theta = abs(dot(last_ray_direction, normal));
                    const auto material_parameters = g_materials[payload.material_index];
                    const auto material = DefaultShading(material_parameters, abs_cos_theta, payload.texcoord);
                    albedo = material.rho(abs_cos_theta);
                }
            }

        } while (payload.bounces < g_camera_state.max_bounce_count && !is_black(payload.throughput));

        return payload.radiance;
    });

    // Accumulate normals
    // TODO Transform to camera space and potentialy flip y/z.
    if (g_AI_denoiser_state.normals_buffer != 0) {
        auto normals_buffer = g_AI_denoiser_state.normals_buffer;
        float3 prev_normals = make_float3(normals_buffer[g_launch_index]);
        const float magnitude = g_camera_state.accumulations ? normals_buffer[g_launch_index].w : 0.0f;
        prev_normals *= magnitude;

        // OptiX expects a normal in view space with red going from left to right, green as up and blue along the depth, with normals pointing towards the camera as 100% blue.
        float3 denoiser_normal = g_camera_state.world_to_view_rotation * normal;
        denoiser_normal.z = -denoiser_normal.z;

        float3 accumulated_normals = prev_normals + denoiser_normal;
        float new_length = length(accumulated_normals);
        normals_buffer[g_launch_index] = make_float4(accumulated_normals / new_length, new_length);
    }

    // Accumulate albedo
    if (g_AI_denoiser_state.albedo_buffer != 0) {
        auto albedo_buffer = g_AI_denoiser_state.albedo_buffer;
        const float3 prev_albedo = make_float3(albedo_buffer[g_launch_index]);
        const float accumulation_count = g_camera_state.accumulations ? (albedo_buffer[g_launch_index].w + 1.0f) : 1.0f;
        const float3 accumulated_albedo = lerp(prev_albedo, albedo, 1.0f / accumulation_count);
        albedo_buffer[g_launch_index] = make_float4(accumulated_albedo, accumulation_count);
    }

    // Output radiance.
#ifdef DOUBLE_PRECISION_ACCUMULATION_BUFFER
    double4 p = g_camera_state.accumulation_buffer[g_launch_index];
    float4 noisy_pixel = make_float4(p.x, p.y, p.z, 1.0f);
#else
    float4 noisy_pixel = g_camera_state.accumulation_buffer[g_launch_index];
#endif

    g_AI_denoiser_state.noisy_pixels_buffer[g_launch_index] = gamma_correct(noisy_pixel);
}

RT_PROGRAM void copy_to_output() {
    float4 pixel = reverse_gamma_correct(g_AI_denoiser_state.denoised_pixels_buffer[g_launch_index]);

    if (g_AI_denoiser_state.flags & unsigned int(AIDenoiserFlag::VisualizeNoise)) {
#ifdef DOUBLE_PRECISION_ACCUMULATION_BUFFER
        double4 p = g_camera_state.accumulation_buffer[g_launch_index];
        pixel = make_float4(p.x, p.y, p.z, 1.0f);
#else
        pixel = g_camera_state.accumulation_buffer[g_launch_index];
#endif
    } else if (g_AI_denoiser_state.flags & int(AIDenoiserFlag::VisualizeAlbedo))
        pixel = g_AI_denoiser_state.albedo_buffer[g_launch_index];
    else if (g_AI_denoiser_state.flags & int(AIDenoiserFlag::VisualizeNormals))
        pixel = g_AI_denoiser_state.normals_buffer[g_launch_index] * 0.5f + 0.5f;

    g_camera_state.output_buffer[g_launch_index] = float_to_half(pixel);
}

} // NS AIDenoiser

//-------------------------------------------------------------------------------------------------
// Ray generation program for visualizing estimated and sampled albedo.
//-------------------------------------------------------------------------------------------------

RT_PROGRAM void albedo_RPG() {

    accumulate([](MonteCarloPayload payload) -> float3 {
        float3 last_ray_direction = payload.direction;
        do {
            last_ray_direction = payload.direction;
            Ray ray(payload.position, payload.direction, RayTypes::MonteCarlo, g_scene.ray_epsilon);
            rtTrace(g_scene_root, ray, payload);
        } while (payload.material_index == 0 && !is_black(payload.throughput));

        size_t2 screen_size = g_camera_state.accumulation_buffer.size();
        bool valid_material = payload.material_index != 0;
        if (g_launch_index.x < screen_size.x / 2 && valid_material) {
            using namespace Shading::ShadingModels;
            const Material& material_parameter = g_materials[payload.material_index];
            const float abs_cos_theta = abs(dot(last_ray_direction, payload.shading_normal));
            const DefaultShading material = DefaultShading(material_parameter, abs_cos_theta, payload.texcoord);
            return material.rho(abs_cos_theta);
        } else
            return payload.throughput;
    });
}

//-------------------------------------------------------------------------------------------------
// Miss program for monte carlo rays.
//-------------------------------------------------------------------------------------------------

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(MonteCarloPayload, monte_carlo_payload, rtPayload, );

RT_PROGRAM void miss() {
    float3 environment_radiance = g_scene.environment_tint;

    unsigned int environment_map_ID = g_scene.environment_light.environment_map_ID;
    if (environment_map_ID) {
        bool next_event_estimated = monte_carlo_payload.bounces != 0; // Was next event estimated at previous intersection.
        environment_radiance *= LightSources::evaluate_intersection(g_scene.environment_light, ray.origin, ray.direction, 
                                                                    monte_carlo_payload.bsdf_MIS_PDF, next_event_estimated);
    }

    monte_carlo_payload.radiance += monte_carlo_payload.throughput * environment_radiance;
    monte_carlo_payload.throughput = make_float3(0.0f);
    monte_carlo_payload.shading_normal = -ray.direction;
}

//-------------------------------------------------------------------------------------------------
// Exception program.
//-------------------------------------------------------------------------------------------------
RT_PROGRAM void exceptions() {
    rtPrintExceptionDetails();
}
